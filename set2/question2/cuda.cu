#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <omp.h>

#define N 5000
#define INDEX(i, j) (i * N + j)

double t1, t2;

double get_wtime()
{
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec + t.tv_usec * 1e-6;
}

void CPUvecAdd(double *a, double *b, double *c)
{
    for (int i = 0; i < N * N; i++)
    {
        c[i] = a[i] + b[i];
    }
}

void CPUvecSub(double *a, double *b, double *c)
{
    for (int i = 0; i < N * N; i++)
    {
        c[i] = a[i] - b[i];
    }
}

void CPUmatMult(double *result, double *a, double *b)
{
    // Zero the result matrix first
    memset(result, 0, N * N * sizeof(double));
    
    // Column-major ordering (j,i,k)
    for (int j = 0; j < N; j++)
    {
        for (int i = 0; i < N; i++)
        {
            for (int k = 0; k < N; k++)
            {
                result[j * N + i] += a[k * N + i] * b[j * N + k];
            }
        }
    }
}

void initializeMatrix(double *matrix, unsigned int seed)
{
#pragma omp for collapse(2)
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int index = INDEX(i, j);
            matrix[index] = (double)rand_r(&seed) / RAND_MAX;
            //matrix[index] = 5; // testing
        }
    }
}

int main(int argc, char **argv)
{
    // Add at start of main:
    hipblasHandle_t handle, handle2;
    hipblasCreate(&handle);
    hipblasCreate(&handle2);

    int numDev;
    hipGetDeviceCount(&numDev);
    if (numDev < 1)
    {
        printf("CUDA device missing! Do you need to use optirun?\n");
        return 1;
    }

    // Use 1D array for N*N matrixes
    // Host
    double *A = (double *)malloc(sizeof(double) * N * N);
    double *B = (double *)malloc(sizeof(double) * N * N);
    double *C = (double *)malloc(sizeof(double) * N * N);
    double *D = (double *)malloc(sizeof(double) * N * N);
    double *E = (double *)malloc(sizeof(double) * N * N);
    double *F = (double *)malloc(sizeof(double) * N * N);
    double *temp = (double *)malloc(sizeof(double) * N * N);

    double *E_CPU = (double *)malloc(sizeof(double) * N * N);
    double *F_CPU = (double *)malloc(sizeof(double) * N * N);

    // random seeds
    int seed[] = {1, 10, 100, 1000};
    initializeMatrix(A, seed[0]);
    initializeMatrix(B, seed[1]);
    initializeMatrix(C, seed[2]);
    initializeMatrix(D, seed[3]);

    // printf("First number of matrix A: %f \n", A[0]);
    // printf("First number of matrix B: %f \n", B[0]);
    // printf("First number of matrix C: %f \n", C[0]);
    // printf("First number of matrix D: %f \n", D[0]);

    // Device
    double *d_A;
    double *d_B;
    double *d_C;
    double *d_D;
    double *d_E;
    double *d_F;

    // remember to attempt allocating memory from gpu see if its faster
    hipMalloc(&d_A, sizeof(double) * N * N);
    hipMalloc(&d_B, sizeof(double) * N * N);
    hipMalloc(&d_C, sizeof(double) * N * N);
    hipMalloc(&d_D, sizeof(double) * N * N);
    hipMalloc(&d_E, sizeof(double) * N * N);
    hipMalloc(&d_F, sizeof(double) * N * N);

    t1 = get_wtime();
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_D, D, N * N * sizeof(double), hipMemcpyHostToDevice);

    const double alpha = 1.0;
    const double beta = 0.0;
    const double reverse = -1.0;


    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_C, N, &beta, d_E, N);
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &reverse, d_B, N, d_D, N, &alpha, d_E, N);


    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_D, N, &beta, d_F, N);
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_C, N, &alpha, d_F, N);

    hipMemcpy(E, d_E, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(F, d_F, N * N * sizeof(double), hipMemcpyDeviceToHost);
    t2 = get_wtime();

    double cuda_time = t2 - t1;
    //printf("CUDA (CUBLAS) Result Matrix E:\n");
    //for (int i = 0; i < N; i++)
    //{
    //    for (int j = 0; j < N; j++)
    //    {
    //        printf("\t%f", E[INDEX(i, j)]);
    //    }
    //    printf("\n");
    //}
    printf("CUDA (CUBLAS): First val = %f, Took %f seconds\n", E[0], cuda_time);
    
    t1 = get_wtime();
    // E = BD - AC
    CPUmatMult(temp, A, C);        // temp = AC
    CPUmatMult(E_CPU, B, D);       // E_CPU = BD
    CPUvecSub(temp, E_CPU, E_CPU); // E_CPU = BD - AC

    // F = AD + BC
    CPUmatMult(temp, A, D);        // temp = AD
    CPUmatMult(F_CPU, B, C);       // F_CPU = BC
    CPUvecAdd(F_CPU, F_CPU, temp); // F_CPU = AD + BC
    t2 = get_wtime();

    double cpu_time = t2 - t1;

    //printf("CPU Result Matrix E:\n");
    //for (int i = 0; i < N; i++)
    //{
    //    for (int j = 0; j < N; j++)
    //    {
    //        printf("\t%f", E_CPU[INDEX(i, j)]);
    //    }
    //    printf("\n");
    //}
    printf("CPU: First val = %f, Took %f seconds\n", E_CPU[0], cpu_time);

    float speedup = cpu_time / cuda_time;

    printf("\nSpeedup: %f\n", speedup);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_E);

    hipblasDestroy(handle);

    free(A);
    free(B);
    free(C);
    free(D);
    free(E);
    free(temp);

    return 0;
}
